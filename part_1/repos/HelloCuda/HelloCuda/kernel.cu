#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0); // CuDa kartes izveele vairaaku karshu sisteemaas
	// kljuuda, ja nav cuda ieriices
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    // Launch a kernel on the GPU with one thread for each element.
	// kodola funkcijas izsaukshana, nav c++ templates, bet cuda specifisks izsaukums, nododam tikai GPU adreses
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

*/

#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h> 
#include <iostream>

// elementu skaits
#define N (2096*4096)
// core skaits
#define M 1024

void random_ints(int *a, int n) {
	srand((unsigned int)time(NULL));

	for (int i = 0; i < n; i++) {
		a[i] = rand();
	}
}

__global__ void add(int *a, int *b, int *c, int n) {
	// paarejam uz masiivu indeksu
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < n)
		c[index] = a[index] + b[index];
}

int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a, b, c
	// and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N blocks
	//thre ads_per_blocks 512/256
	// blokus kaits peec datiem massiivaa
	// cuda toolkit ekselis paliidz apreekinaat efektiivos parametrus
	add << <(N + M - 1) / M, M >> > (d_a, d_b, d_c, N);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}

// piemeers ar datu kopeeshanu keshatminjaa apstraadei
__global__ void stencil_1d(int *in, int *out) {
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;
	// Read input elements into shared memory
	// Katrs kodols aizpilda vienu elementu (nav cikla)
	temp[lindex] = in[gindex];
	if (threadIdx.x < RADIUS) {
		temp[lindex - RADIUS] = in[gindex - RADIUS];
		temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
	}

	// Sheit vajadziigs __synctrheads(), lai paarliecinaatos, ka visa atminja ir nokopeeta

	// Apply the stencil
	int result = 0;
	for (int offset = -RADIUS; offset <= RADIUS; offset++)
		result += temp[lindex + offset];
	// Store the result
	out[gindex] = result;
}