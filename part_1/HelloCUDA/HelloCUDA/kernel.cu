#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h> 
#include <iostream>

#define N (2096*4096)
#define M 1024

void random_ints(int *a, int n) {
	srand((unsigned int)time(NULL));
	
	for (int i = 0; i < n; i++) {
		a[i] = rand();
	}
}

__global__ void add(int *a, int *b, int *c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (index < n)
		c[index] = a[index] + b[index];
}

int main(void) {
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; // device copies of a, b, c
	int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Alloc space for host copies of a, b, c
	// and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU with N blocks
	add<<<(N + M - 1) / M, M>>>(d_a, d_b, d_c, N);
	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}

