#include "hip/hip_runtime.h"

#include <hiprand/hiprand_kernel.h>

#include <hiprand/hiprand.h>

#include <stdio.h>
#include <time.h>
#include <assert.h>
#include <iostream>

using namespace std;

void generateRandomCudaHost(float* localMemRandom, unsigned int arraySize, unsigned int memsize);

void checkCUDAError(const char* msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(-1);
	}
}

__global__ void add(int* a, int* b, int* c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < n)
		c[index] = a[index] + b[index];
}

int main()
{
	float* localMemRandom;
	unsigned int arraySize = 1024 * 1024 * 100;
	size_t memsize = sizeof(float) * arraySize;
	localMemRandom = (float*)malloc(memsize);

	// Generate random and copy to local memory.
	generateRandomCudaHost(localMemRandom, arraySize, memsize);

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Reduce elements to average

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "---------------------" << endl;
	cout << "random generation time for size " << (memsize / 1024 / 1024) << "MB  on GPU: " << elapsedTime << " milliseconds" << endl;
	cout << "---------------------" << endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();
	checkCUDAError("cuda device reset");

	return 0;
}


void generateRandomCudaHost(float* localMemRandom, unsigned int arraySize, unsigned int memsize) {

	checkCUDAError("memory allocation");

	hiprandGenerator_t randomGenerator;
	hiprandCreateGeneratorHost(&randomGenerator, HIPRAND_RNG_PSEUDO_DEFAULT);
	checkCUDAError("random generator creation");

	hiprandSetPseudoRandomGeneratorSeed(randomGenerator, time(NULL));
	checkCUDAError("seeding random generator");

	hiprandGenerateUniform(randomGenerator, localMemRandom, arraySize);
	checkCUDAError("random generation process");

	hiprandDestroyGenerator(randomGenerator);
	checkCUDAError("freeing generator memory");
}